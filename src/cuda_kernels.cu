#include "hip/hip_runtime.h"
#include "cuda_kernels.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void matrix_multiply_kernel(double* d_c, const double* d_a, const double* d_b, int a_rows, int a_cols, int b_cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < a_rows && col < b_cols) {
        double sum = 0.0;
        for (int k = 0; k < a_cols; ++k) {
            sum += d_a[row * a_cols + k] * d_b[k * b_cols + col];
        }
        d_c[row * b_cols + col] = sum;
    }
}

#define CUDA_CHECK(err) __cuda_check_error(err, __FILE__, __LINE__)

static void __cuda_check_error(hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error at %s:%d: %s\n", file, line, hipGetErrorString(err));
    }
}

int matrix_multiply_gpu(Matrix* c, const Matrix* a, const Matrix* b) {
    double *d_a = NULL, *d_b = NULL, *d_c = NULL;
    hipError_t err;

    size_t size_a = a->rows * a->cols * sizeof(double);
    err = hipMalloc((void**)&d_a, size_a);
    if (err != hipSuccess) { CUDA_CHECK(err); goto cleanup; }

    size_t size_b = b->rows * b->cols * sizeof(double);
    err = hipMalloc((void**)&d_b, size_b);
    if (err != hipSuccess) { CUDA_CHECK(err); goto cleanup; }

    size_t size_c = c->rows * c->cols * sizeof(double);
    err = hipMalloc((void**)&d_c, size_c);
    if (err != hipSuccess) { CUDA_CHECK(err); goto cleanup; }


    err = hipMemcpy(d_a, a->data, size_a, hipMemcpyHostToDevice);
    if (err != hipSuccess) { CUDA_CHECK(err); goto cleanup; }

    err = hipMemcpy(d_b, b->data, size_b, hipMemcpyHostToDevice);
    if (err != hipSuccess) { CUDA_CHECK(err); goto cleanup; }

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((b->cols + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (a->rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrix_multiply_kernel<<<numBlocks, threadsPerBlock>>>(d_c, d_a, d_b, a->rows, a->cols, b->cols);
    CUDA_CHECK(hipGetLastError());

    err = hipMemcpy(c->data, d_c, size_c, hipMemcpyDeviceToHost);
    if (err != hipSuccess) { CUDA_CHECK(err); goto cleanup; }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;

cleanup:
    if (d_a) hipFree(d_a);
    if (d_b) hipFree(d_b);
    if (d_c) hipFree(d_c);
    return 1;
}
